#include<iostream>
#include<fstream>
#include<string>
#include<cmath>
#include<assert.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
//using namespace std;
typedef double real;

std::string part_input_file,part_out_name_base,vtk_out_name_base;

real timestep_length,time_end,epsilon,sigma;

int part_out_freq,vtk_out_freq,cl_workgroup_1dsize;

void checkError (hipError_t err)
{
    if(err != hipSuccess )
    {
        std::cout<< hipGetErrorString(err) <<std::endl ;
        exit(-1);
    }
}

__host__ void fileread(std::string file);
//void force_update(real* pos_x, real* pos_y, real* pos_z, real* F_x, real* F_y, real* F_z, real sigma, real epsilon, unsigned int N);
////////////////////////////*************************************************************************************************************************************
__global__ void Force_update_1D(
		real *d_pos_x,
		real *d_pos_y,
		real *d_pos_z,
		real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real sigma,
		real epsilon,
		unsigned int N)
{


	
	
    	real d = 0.0,
            d_2 = 0.0,
            x_i = 0.0,
            y_i = 0.0,
            dx,dy,dz,
            z_i = 0.0,
            c1 = 0.0,
            t_pow = 0.0,
            sig_abs = 0.0,
            tempx = 0.0,
            tempy = 0.0,
            tempz = 0.0;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
    	c1 = 24 * epsilon;
    
            x_i = d_pos_x[id];
            y_i = d_pos_y[id];
            z_i = d_pos_z[id];
            
            
              for(auto j=0;j<N;++j){
                            
                            if(id != j)
                            {
                                
                                d_2 = (x_i - d_pos_x[j]) * (x_i - d_pos_x[j]) + (y_i - d_pos_y[j])*(y_i - d_pos_y[j]) + (z_i - d_pos_z[j]) * (z_i - d_pos_z[j]);
                                //d = (x_i - pos_x[j])  + (y_i - pos_y[j]) + (z_i - pos_z[j]);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"d: "<< d <<"\n"; 
                                d = sqrt(d_2);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"d_2: "<< d_2 <<"\n"; 
                                //abs_d = fabs(d);
                                dx = x_i - d_pos_x[j];
                                dy = y_i - d_pos_y[j];
                                dz = z_i - d_pos_z[j];
                                //std::cout<< i<<"\t" <<j<<"\t"<<"abs_d: "<< abs_d <<"\n"; 
                                assert(d != 0);
                                sig_abs = sigma/d;
                                t_pow = pow(sig_abs,6);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"weird calc: "<< ((c1/(d_2) * t_pow * (2*t_pow - 1)) * d) <<"\t" << "c1: " << c1<<"\n"; 
                                tempx = tempx + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dx); 
                                tempy = tempy + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dy); 
                                tempz = tempz + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dz); 
                                //std::cout<< i<<"\t" <<j<<"\t"<<"temp: "<< temp <<"\n";
                            }
                                
                                
                   }
               d_F_x[id] = tempx; 
               d_F_y[id] = tempy; 
               d_F_z[id] = tempz; 
               //std::cout <<"updated F: " << i << "\t" << F[i]<<"\n";
               /*tempx = 0;
               tempy = 0;
               tempz = 0;*/
                            
}


__global__ void pos_update_1D(
		real *d_pos_x,
		real *d_pos_y,
		real *d_pos_z,
		real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real *mass,
		real timestep_length){

			int i = threadIdx.x + blockDim.x * blockIdx.x;
		        real del_T = timestep_length*timestep_length;
                            d_pos_x[i] = d_pos_x[i] + timestep_length * (d_vel_x[i]) + ((del_T/(2*mass[i])) * (d_F_x[i]));
                            d_pos_y[i] = d_pos_y[i] + timestep_length * (d_vel_y[i]) + ((del_T/(2*mass[i])) * (d_F_y[i]));
                            d_pos_z[i] = d_pos_z[i] + timestep_length * (d_vel_z[i]) + ((del_T/(2*mass[i])) * (d_F_z[i]));
                            //std::cout << i <<"\t" << pos_x[i] << "\t" << pos_y[i] << "\t" << pos_z[i] <<"\n";
                            d_F_old_x[i] = d_F_x[i];
                            d_F_old_y[i] = d_F_y[i];
                            d_F_old_z[i] = d_F_z[i];

}
__global__ void vel_update_1D(
			real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real timestep_length,
		real *mass){

		int i = threadIdx.x + blockDim.x * blockIdx.x;
		d_vel_x[i] = d_vel_x[i] + timestep_length * 0.5 * (d_F_x[i] + d_F_old_x[i])/mass[i];
                d_vel_y[i] = d_vel_y[i] + timestep_length * 0.5 * (d_F_y[i] + d_F_old_y[i])/mass[i];
                d_vel_z[i] = d_vel_z[i] + timestep_length * 0.5 * (d_F_z[i] + d_F_old_z[i])/mass[i];

}







////////////////////**********************************************************************************************************************************************
int main(int argc,char *argv[])
{
        std::cout.precision(4);
        std::string para_file;
        if(argc==1)
        {
            std::cout<< "Para file name not given" << std::endl;
            return 0;
        }
        para_file = argv[1];
        //std::cout<< para_file<< " " << argc << std::endl; //parameter file name

        fileread(para_file); //rad parameter file

        unsigned int N=0; //Number of particles 
        //*********************************************************************
        ///Read from parameter file
        //*********************************************************************
        std::ifstream f;
        f.open(part_input_file); //read input file
        f>>N;
        //std::cout<< "Number of particles: " << N << std::endl;

        real *mass  = new real[N];
	//real *d_mass;
        real *pos_x = new real[N];
	real *d_pos_x;
        real *pos_y = new real[N];
	real *d_pos_y;
        real *pos_z = new real[N];
	real *d_pos_z;
        real *vel_x = new real[N];
	real *d_vel_x;
        real *vel_y = new real[N];
	real *d_vel_y;
        real *vel_z = new real[N];
	real *d_vel_z;
                
        real *F_x = new real[N];
        real *d_F_x;
        real *F_y = new real[N];
        real *d_F_y;
        real *F_z = new real[N];
        real *d_F_z;
        real *F_old_x = new real[N];
	real *d_F_old_x;
        real *F_old_y = new real[N];
	real *d_F_old_y;
        real *F_old_z = new real[N];
	real *d_F_old_z;
        int count = 0;

        /*d: particle distance between ith and jth particle
        //d_2: sqare of d.
        //abs_d_x: absolute of (d_x)
        //t: time step increment
        //temp: for reduction of force
	****************************************************/
        
        real t = 0;

        int iter =0;
        while (true) {
            f >> mass[iter] >> pos_x[iter] >> pos_y[iter]>> pos_z[iter] >> vel_x[iter] >> vel_y[iter] >> vel_z[iter];
            if( f.eof() ) break;
            //std::cout<< mass[i] << " " <<  pos_x[i] <<" "<< pos_y[i] << " " << pos_z[i] << " " << vel_x[i] << " " << vel_y[i] <<" " << vel_z[i]<< std::endl;
            ++iter;
        }
        f.close();
        //*********************************************************************
        //*********************************************************************
        //*********************************************************************
        std::string vtk_file =" ";
        std::ofstream vtk;
	//del_T: del T square

        

	/*First force kernel cal using cuda*/

	//Cuda memory assignment:
	hipMalloc((void**)&d_pos_x, N*sizeof(real));
	hipMalloc((void**)&d_pos_y, N*sizeof(real));
	hipMalloc((void**)&d_pos_z, N*sizeof(real));
	hipMalloc((void**)&d_vel_x, N*sizeof(real));
	hipMalloc((void**)&d_vel_y, N*sizeof(real));
	hipMalloc((void**)&d_vel_z, N*sizeof(real));
	hipMalloc((void**)&d_F_x, N*sizeof(real));
	hipMalloc((void**)&d_F_y, N*sizeof(real));
	hipMalloc((void**)&d_F_z, N*sizeof(real));
	hipMalloc((void**)&d_F_old_x, N*sizeof(real));
	hipMalloc((void**)&d_F_old_y, N*sizeof(real));
	hipMalloc((void**)&d_F_old_z, N*sizeof(real));

	/**********Memcpy for initiaized variables ******************/
	hipMemcpy(d_pos_x,pos_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_pos_y,pos_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_pos_z,pos_z, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_vel_x,vel_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_vel_y,vel_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_vel_z,vel_z, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_x,F_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_y,F_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_z,F_z, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old_x,F_old_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old_y,F_old_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old_z,F_old_z, (N*sizeof(real)),hipMemcpyHostToDevice);

	//real t = 0;


	
	


	Force_update_1D<<<1,N>>>(d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,sigma,epsilon,N);
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
 		 printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
  	printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
         do{
                
                    //position update and parallely copy force to force_old
                     pos_update_1D<<<1,N>>>(d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,mass,timestep_length);       

			
                    //Force update
                           //__syncAllThreads
			Force_update_1D<<<1,N>>>(d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,sigma,epsilon,N);
			hipDeviceSynchronize();
			//__syncthreads();
                            //__synchAllThreads    
                    //calculate velocity 
                      vel_update_1D<<<1,N>>>(d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,timestep_length,mass);      
                            
        hipMemcpy(pos_x,d_pos_x, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(pos_y,d_pos_y, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(pos_z,d_pos_z, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(vel_x,d_vel_x, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(vel_y,d_vel_y, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(vel_z,d_vel_z, (N*sizeof(real)),hipMemcpyDeviceToHost);    
                        
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                        vtk_file  = "tmp/" + vtk_out_name_base + std::to_string(count) +".vtk";
                //        std::cout << vtk_file << std::endl;
                        vtk.open(vtk_file);
                        vtk << "# vtk DataFile Version 4.0" << "\n" << "hesp visualization file" << "\n" << "ASCII" << "\n" << "DATASET UNSTRUCTURED_GRID" << "\n" << "POINTS 2 double" << "\n";
                        vtk<< std::fixed;
                        for(int j =0; j<N; ++j)
                            vtk<<pos_x[j] << " "<< pos_y[j]<< " " << pos_z[j] << "\n";
                        vtk << "CELLS 0 0" << "\n";
                        vtk << "CELL_TYPES 0" << "\n";
                        vtk << "POINT_DATA 2" << "\n";
                        vtk << "SCALARS m double" << "\n";
                        vtk << "LOOKUP_TABLE default" << "\n";
                        vtk<< std::fixed;
                        for(int j =0; j<N; ++j)
                            vtk<< mass[j] << "\n";
                        vtk <<"VECTORS v double" << "\n";
                        vtk<<  std::fixed;
                        for(int j =0; j<N; ++j)
                            vtk<<vel_x[j] << " "<< vel_y[j]<< " " << vel_y[j] << "\n";
                        vtk.close();
                        vtk_file =" ";
                            
                            
                            
                            count++;
                            t = t + 0.01;
            }while(t<time_end);
                        

        delete(mass);
        delete(pos_x);
        delete(pos_y);
        delete(pos_z);
        delete(vel_x);
        delete(vel_y);
        delete(vel_z);
        delete(F_x);
        delete(F_y);
        delete(F_z);
        delete(F_old_x);
        delete(F_old_y);
        delete(F_old_z);
	/*delete(d_pos_x);
        delete(d_pos_y);
        delete(d_pos_z);
        delete(d_vel_x);
        delete(d_vel_y);
        delete(d_vel_z);
        delete(d_F_x);
        delete(d_F_y);
        delete(d_F_z);
        delete(d_F_old_x);
        delete(d_F_old_y);
        delete(d_F_old_z);*/


        return 0;
}

       

void fileread(std::string file){
    std::ifstream ff;
    std::string para_name;
    std::string value;
    ff.open(file);
    std::string::size_type sz; 
    for(int i =0; i<10; ++i)
    {
        ff>>para_name >> value;
        if(para_name=="part_input_file"){
            part_input_file = value;
            //std::cout<< "part_input_file " << part_input_file<< std::endl;
        }
        else if(para_name=="timestep_length"){
            timestep_length= std::stod  (value,&sz);
            //std::cout<< "timestep_length " << timestep_length<< std::endl;
        }
        else if(para_name=="time_end"){
           time_end= std::stod  (value,&sz);
            //std::cout<< "time_end " << time_end<< std::endl; 
        }
        else if (para_name=="sigma"){
            sigma=std::stod  (value,&sz);
            //std::cout<< "sigma " << sigma<< std::endl;
        }
        else if (para_name=="epsilon"){
            epsilon=std::stod  (value,&sz);;
            //std::cout<< "epsilon " << epsilon<< std::endl;
        }
        else if (para_name=="part_out_freq"){
            part_out_freq=std::stoi  (value,&sz);
            //std::cout<< "part_out_freq " << part_out_freq<< std::endl;
            
        }
           else if (para_name=="part_out_name_base"){
            part_out_name_base=value;
            //std::cout<< "part_out_name_base " << part_out_name_base<< std::endl;
               
        }
        else if (para_name=="vtk_out_freq")
        {
            vtk_out_freq=std::stoi  (value,&sz);
            //std::cout<< "vtk_out_freq " << vtk_out_freq<< std::endl;
        }
        else if (para_name=="vtk_out_name_base"){
            vtk_out_name_base=value;
            //std::cout<< "vtk_out_name_base " << vtk_out_name_base<< std::endl;
        }
        else if (para_name=="cl_workgroup_1dsize"){
            cl_workgroup_1dsize=std::stoi  (value,&sz);
            //std::cout<< "cl_workgroup_1dsize " << cl_workgroup_1dsize<< std::endl;
        }
    }
    ff.close();
}
