#include<iostream>
#include<fstream>
#include<string>
#include<cmath>
#include<assert.h>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
//using namespace std;

double getSeconds()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

typedef double real;

std::string part_input_file,part_out_name_base,vtk_out_name_base;

real timestep_length,time_end,epsilon,sigma;

int part_out_freq,vtk_out_freq,cl_workgroup_1dsize;

void checkError (hipError_t err)
{
    if(err != hipSuccess )
    {
        std::cout<< hipGetErrorString(err) <<std::endl ;
        exit(-1);
    }
}

__host__ void fileread(std::string file);
//void force_update(real* pos_x, real* pos_y, real* pos_z, real* F_x, real* F_y, real* F_z, real sigma, real epsilon, unsigned int N);
////////////////////////////*************************************************************************************************************************************
__global__ void Force_update_1D(
		real *d_pos_x,
		real *d_pos_y,
		real *d_pos_z,
		real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real sigma,
		real epsilon,
		unsigned int N)
{


	
	
    	real d = 0.0,
            d_2 = 0.0,
            x_i = 0.0,
            y_i = 0.0,
            dx,dy,dz,
            z_i = 0.0,
            c1 = 0.0,
            t_pow = 0.0,
            sig_abs = 0.0,
            tempx = 0.0,
            tempy = 0.0,
            tempz = 0.0;
            
            
	int id = threadIdx.x + blockIdx.x*blockDim.x;
    
    c1 = 24 * epsilon;
    
            x_i = d_pos_x[id];
            y_i = d_pos_y[id];
            z_i = d_pos_z[id];
            
            
              for(auto j=0;j<N;++j){
                            
                            if(id != j)
                            {
                                
                                d_2 = (x_i - d_pos_x[j]) * (x_i - d_pos_x[j]) + (y_i - d_pos_y[j])*(y_i - d_pos_y[j]) + (z_i - d_pos_z[j]) * (z_i - d_pos_z[j]);
                                d = sqrt(d_2);
                                dx = x_i - d_pos_x[j];
                                dy = y_i - d_pos_y[j];
                                dz = z_i - d_pos_z[j];
                                assert(d != 0);
                                sig_abs = sigma/d;
                                t_pow = pow(sig_abs,6);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"weird calc: "<< ((c1/(d_2) * t_pow * (2*t_pow - 1)) * d) <<"\t" << "c1: " << c1<<"\n"; 
                                tempx = tempx + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dx); 
                                tempy = tempy + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dy); 
                                tempz = tempz + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dz); 
                                //std::cout<< id<<"\t" <<j<<"\t"<<"temp: "<< temp <<"\n";
                            }
                                
                                
                   }
            
               d_F_x[id] = tempx; 
               d_F_y[id] = tempy; 
               d_F_z[id] = tempz; 
              
                            
}


__global__ void pos_update_1D(
		real *d_pos_x,
		real *d_pos_y,
		real *d_pos_z,
		real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real *mass,
		real timestep_length){

			int i = threadIdx.x + blockDim.x * blockIdx.x;
                //printf("%d\n",i);
		        real del_T = timestep_length*timestep_length;
                //printf("%f\t%f\t%f\n",mass[i],d_vel_x[i],d_F_x[i]);
                            d_pos_x[i] = d_pos_x[i] + timestep_length * (d_vel_x[i]) + ((del_T/(2*mass[i])) * (d_F_x[i]));
                            d_pos_y[i] = d_pos_y[i] + timestep_length * (d_vel_y[i]) + ((del_T/(2*mass[i])) * (d_F_y[i]));
                            d_pos_z[i] = d_pos_z[i] + timestep_length * (d_vel_z[i]) + ((del_T/(2*mass[i])) * (d_F_z[i]));
                            //std::cout << i <<"\t" << pos_x[i] << "\t" << pos_y[i] << "\t" << pos_z[i] <<"\n";
                            d_F_old_x[i] = d_F_x[i];
                            d_F_old_y[i] = d_F_y[i];
                            d_F_old_z[i] = d_F_z[i];

}
__global__ void vel_update_1D(
			real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real timestep_length,
		real *mass){

		int i = threadIdx.x + blockDim.x * blockIdx.x;
		d_vel_x[i] = d_vel_x[i] + timestep_length * 0.5 * (d_F_x[i] + d_F_old_x[i])/mass[i];
                d_vel_y[i] = d_vel_y[i] + timestep_length * 0.5 * (d_F_y[i] + d_F_old_y[i])/mass[i];
                d_vel_z[i] = d_vel_z[i] + timestep_length * 0.5 * (d_F_z[i] + d_F_old_z[i])/mass[i];

}
 __global__ void print(real *d_pos_x, unsigned int N){
    
        int i = threadIdx.x;
            printf("%f\n",d_pos_x[i]);
}






////////////////////**********************************************************************************************************************************************
int main(int argc,char *argv[])
{
        std::cout.precision(4);
        std::string para_file;
        if(argc==1)
        {
            std::cout<< "Para file name not given" << std::endl;
            return 0;
        }
        para_file = argv[1];
        //std::cout<< para_file<< " " << argc << std::endl; //parameter file name
        double wcTimeStart= 0.0, wcTimeEnd=0.0;

        fileread(para_file); //rad parameter file

        unsigned int N=0; //Number of particles 
        //*********************************************************************
        ///Read from parameter file
        //*********************************************************************
        std::ifstream f;
        f.open(part_input_file); //read input file
        f>>N;
        //std::cout<< "Number of particles: " << N << std::endl;

        real *mass  = new real[N];
	real *d_mass;
        real *pos_x = new real[N];
	    real *d_pos_x;
        real *pos_y = new real[N];
        real *d_pos_y;
        real *pos_z = new real[N];
	    real *d_pos_z;
        real *vel_x = new real[N];
	    real *d_vel_x;
        real *vel_y = new real[N];
	    real *d_vel_y;
        real *vel_z = new real[N];
        real *d_vel_z;
               
        real *F_x = new real[N];
        real *d_F_x;
        real *F_y = new real[N];
        real *d_F_y;
        real *F_z = new real[N];
        real *d_F_z;
        real *F_old_x = new real[N];
        real *d_F_old_x;
        real *F_old_y = new real[N];
        real *d_F_old_y;
        real *F_old_z = new real[N];
        real *d_F_old_z;
        int count = 0;
//***************************************
        //Cuda memory assignment:
	hipMalloc((void**)&d_mass, N*sizeof(real));
        hipMalloc((void**)&d_pos_x, N*sizeof(real));
	hipMalloc((void**)&d_pos_y, N*sizeof(real));
	hipMalloc((void**)&d_pos_z, N*sizeof(real));
	hipMalloc((void**)&d_vel_x, N*sizeof(real));
	hipMalloc((void**)&d_vel_y, N*sizeof(real));
	hipMalloc((void**)&d_vel_z, N*sizeof(real));
	hipMalloc((void**)&d_F_x, N*sizeof(real));
	hipMalloc((void**)&d_F_y, N*sizeof(real));
	hipMalloc((void**)&d_F_z, N*sizeof(real));
	hipMalloc((void**)&d_F_old_x, N*sizeof(real));
	hipMalloc((void**)&d_F_old_y, N*sizeof(real));
	hipMalloc((void**)&d_F_old_z, N*sizeof(real));
//***************************************
        /*d: particle distance between ith and jth particle
        //d_2: sqare of d.
        //abs_d_x: absolute of (d_x)
        //t: time step increment
        //temp: for reduction of force
	****************************************************/
        
        real t = 0;

        int iter =0;
        while (true) {
            f >> mass[iter] >> pos_x[iter] >> pos_y[iter]>> pos_z[iter] >> vel_x[iter] >> vel_y[iter] >> vel_z[iter];
            if( f.eof() ) break;
            //std::cout<< mass[iter] << " " <<  pos_x[iter] <<" "<< pos_y[iter] << " " << pos_z[iter] << " " << vel_x[iter] << " " << vel_y[iter] <<" " << vel_z[iter]<< std::endl;
            ++iter;
        }
        f.close();
        //*********************************************************************
        //*********************************************************************
        //*********************************************************************
        std::string vtk_file =" ";
        std::ofstream vtk;
	//del_T: del T square

        

	/*First force kernel cal using cuda*/

	

	/**********Memcpy for initiaized variables ******************/
hipMemcpy(d_mass,mass, (N*sizeof(real)),hipMemcpyHostToDevice);
    hipMemcpy(d_pos_x,pos_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_pos_y,pos_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_pos_z,pos_z, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_vel_x,vel_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_vel_y,vel_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_vel_z,vel_z, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_x,F_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_y,F_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_z,F_z, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old_x,F_old_x, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old_y,F_old_y, (N*sizeof(real)),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old_z,F_old_z, (N*sizeof(real)),hipMemcpyHostToDevice);

	//real t = 0;

    //std::cout<<d_pos_x[0]<<std::endl;
	
	//print<<<1,N>>>(d_vel_z,N);

    wcTimeStart = getSeconds(); //Start time
	Force_update_1D<<<1,N>>>(d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,sigma,epsilon,N);
    
	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess) 
 		 printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
  	printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
    
    
    hipDeviceSynchronize();
    //std::cout<<"kernel op after force update"<<std::endl;
    //print<<<1,N>>>(d_F_x,N);
         do{
                //std::cout<<"into the do loop"<<std::endl;
                    //position update and parallely copy force to force_old
                     pos_update_1D<<<1,N>>>(d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,d_mass,timestep_length);       
                    
			
                    //Force update
                           //__syncAllThreads
			Force_update_1D<<<1,N>>>(d_pos_x, d_pos_y, d_pos_z, d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,sigma,epsilon,N);
			hipDeviceSynchronize();
			//__syncthreads();
                            //__synchAllThreads    
                    //calculate velocity 
                      vel_update_1D<<<1,N>>>(d_vel_x, d_vel_y, d_vel_z, d_F_x, d_F_y, d_F_z,d_F_old_x,d_F_old_y,d_F_old_z,timestep_length,d_mass);      
                            
        hipMemcpy(pos_x,d_pos_x, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(pos_y,d_pos_y, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(pos_z,d_pos_z, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(vel_x,d_vel_x, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(vel_y,d_vel_y, (N*sizeof(real)),hipMemcpyDeviceToHost);
	hipMemcpy(vel_z,d_vel_z, (N*sizeof(real)),hipMemcpyDeviceToHost);    
                        
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
                        vtk_file  = "tmp/" + vtk_out_name_base + std::to_string(count) +".vtk";
                //        std::cout << vtk_file << std::endl;
                        vtk.open(vtk_file);
                        vtk << "# vtk DataFile Version 4.0" << "\n" << "hesp visualization file" << "\n" << "ASCII" << "\n" << "DATASET UNSTRUCTURED_GRID" << "\n" << "POINTS "<<N<<" double" << "\n";
                        vtk<< std::fixed;
                        for(int j =0; j<N; ++j)
                            vtk<<pos_x[j] << " "<< pos_y[j]<< " " << pos_z[j] << "\n";
                        vtk << "CELLS 0 0" << "\n";
                        vtk << "CELL_TYPES 0" << "\n";
                        vtk << "POINT_DATA "<< N<< "\n";
                        vtk << "SCALARS m double" << "\n";
                        vtk << "LOOKUP_TABLE default" << "\n";
                        vtk<< std::fixed;
                        for(int j =0; j<N; ++j)
                            vtk<< mass[j] << "\n";
                        vtk <<"VECTORS v double" << "\n";
                        vtk<<  std::fixed;
                        for(int j =0; j<N; ++j)
                            vtk<<vel_x[j] << " "<< vel_y[j]<< " " << vel_y[j] << "\n";
                        vtk.close();
                        vtk_file =" ";
                            
                            
                            
                            count++;
                            t = t + 0.01;
            }while(t<time_end);
        wcTimeEnd = getSeconds(); //End time
        std::cout << "Time Taken for computation: " << wcTimeEnd-wcTimeStart << " sec" << std::endl;              
        
        delete(mass);
        delete(pos_x);
        delete(pos_y);
        delete(pos_z);
        delete(vel_x);
        delete(vel_y);
        delete(vel_z);
        delete(F_x);
        delete(F_y);
        delete(F_z);
        delete(F_old_x);
        delete(F_old_y);
        delete(F_old_z);
	
        hipFree(d_pos_x);
        hipFree(d_pos_y);
        hipFree(d_pos_z);
        hipFree(d_vel_x);
        hipFree(d_vel_y);
        hipFree(d_vel_z);
        hipFree(d_F_x);
        hipFree(d_F_y);
        hipFree(d_F_z);
        hipFree(d_F_old_x);
        hipFree(d_F_old_y);
        hipFree(d_F_old_z);

        return 0;
}

       

void fileread(std::string file){
    std::ifstream ff;
    std::string para_name;
    std::string value;
    ff.open(file);
    std::string::size_type sz; 
    for(int i =0; i<10; ++i)
    {
        ff>>para_name >> value;
        if(para_name=="part_input_file"){
            part_input_file = value;
            //std::cout<< "part_input_file " << part_input_file<< std::endl;
        }
        else if(para_name=="timestep_length"){
            timestep_length= std::stod  (value,&sz);
            //std::cout<< "timestep_length " << timestep_length<< std::endl;
        }
        else if(para_name=="time_end"){
           time_end= std::stod  (value,&sz);
            //std::cout<< "time_end " << time_end<< std::endl; 
        }
        else if (para_name=="sigma"){
            sigma=std::stod  (value,&sz);
            //std::cout<< "sigma " << sigma<< std::endl;
        }
        else if (para_name=="epsilon"){
            epsilon=std::stod  (value,&sz);;
            //std::cout<< "epsilon " << epsilon<< std::endl;
        }
        else if (para_name=="part_out_freq"){
            part_out_freq=std::stoi  (value,&sz);
            //std::cout<< "part_out_freq " << part_out_freq<< std::endl;
            
        }
           else if (para_name=="part_out_name_base"){
            part_out_name_base=value;
            //std::cout<< "part_out_name_base " << part_out_name_base<< std::endl;
               
        }
        else if (para_name=="vtk_out_freq")
        {
            vtk_out_freq=std::stoi  (value,&sz);
            //std::cout<< "vtk_out_freq " << vtk_out_freq<< std::endl;
        }
        else if (para_name=="vtk_out_name_base"){
            vtk_out_name_base=value;
            //std::cout<< "vtk_out_name_base " << vtk_out_name_base<< std::endl;
        }
        else if (para_name=="cl_workgroup_1dsize"){
            cl_workgroup_1dsize=std::stoi  (value,&sz);
            //std::cout<< "cl_workgroup_1dsize " << cl_workgroup_1dsize<< std::endl;
        }
    }
    ff.close();
}
