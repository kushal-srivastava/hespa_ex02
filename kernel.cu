#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
typedef double real;

__global__ void Force_1D(type_T h,
		real *d_pos_x,
		real *d_pos_y,
		real *d_pos_z,
		real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real sigma,
		real epsilon,
		unsigned int N)
{


	
	
    	real d = 0.0,
            d_2 = 0.0,
            abs_d = 0.0,
            x_i = 0.0,
            y_i = 0.0,
            dx,dy,dz,
            z_i = 0.0,
            c1 = 0.0,
            t_pow = 0.0,
            sig_abs = 0.0,
            tempx = 0.0,
            tempy = 0.0,
            tempz = 0.0;
	int id = threadIdx.x + blockIdx.x*blockDim.x;
    	c1 = 24 * epsilon;
    
            x_i = d_pos_x[id];
            y_i = d_pos_y[id];
            z_i = d_pos_z[id];
            
            
              for(auto j=0;j<N;++j){
                            
                            if(i != j)
                            {
                                
                                d_2 = (x_i - d_pos_x[j]) * (x_i - d_pos_x[j]) + (y_i - d_pos_y[j])*(y_i - d_pos_y[j]) + (z_i - d_pos_z[j]) * (z_i - d_pos_z[j]);
                                //d = (x_i - pos_x[j])  + (y_i - pos_y[j]) + (z_i - pos_z[j]);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"d: "<< d <<"\n"; 
                                d = sqrt(d_2);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"d_2: "<< d_2 <<"\n"; 
                                //abs_d = fabs(d);
                                dx = x_i - d_pos_x[j];
                                dy = y_i - d_pos_y[j];
                                dz = z_i - d_pos_z[j];
                                //std::cout<< i<<"\t" <<j<<"\t"<<"abs_d: "<< abs_d <<"\n"; 
                                assert(d != 0);
                                sig_abs = sigma/d;
                                t_pow = pow(sig_abs,6);
                                //std::cout<< i<<"\t" <<j<<"\t"<<"weird calc: "<< ((c1/(d_2) * t_pow * (2*t_pow - 1)) * d) <<"\t" << "c1: " << c1<<"\n"; 
                                tempx = tempx + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dx); 
                                tempy = tempy + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dy); 
                                tempz = tempz + ((c1/(d_2) * t_pow * (2*t_pow - 1)) * dz); 
                                //std::cout<< i<<"\t" <<j<<"\t"<<"temp: "<< temp <<"\n";
                            }
                                
                                
                   }
               d_F_x[i] = tempx; 
               d_F_y[i] = tempy; 
               d_F_z[i] = tempz; 
               //std::cout <<"updated F: " << i << "\t" << F[i]<<"\n";
               /*tempx = 0;
               tempy = 0;
               tempz = 0;*/
                            
}


__global__ void pos_update_1D(
		real *d_pos_x,
		real *d_pos_y,
		real *d_pos_z,
		real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real timestep_length){

			int i = threadIdx.x + blockDim.x * blockIdx.x;
		        real del_T = timestep_length*timestep_length;
                            d_pos_x[i] = d_pos_x[i] + timestep_length * (d_vel_x[i]) + ((del_T/(2*mass[i])) * (d_F_x[i]));
                            d_pos_y[i] = d_pos_y[i] + timestep_length * (d_vel_y[i]) + ((del_T/(2*mass[i])) * (d_F_y[i]));
                            d_pos_z[i] = d_pos_z[i] + timestep_length * (d_vel_z[i]) + ((del_T/(2*mass[i])) * (d_F_z[i]));
                            //std::cout << i <<"\t" << pos_x[i] << "\t" << pos_y[i] << "\t" << pos_z[i] <<"\n";
                            d_F_old_x[i] = d_F_x[i];
                            d_F_old_y[i] = d_F_y[i];
                            d_F_old_z[i] = d_F_z[i];

}

__global__ void vel_update_1D(
			real *d_vel_x,
		real *d_vel_y,
		real *d_vel_z,
		real *d_F_x,
		real *d_F_y,
		real *d_F_z,
		real *d_F_old_x,
		real *d_F_old_y,
		real *d_F_old_z,
		real timestep_length
		real *mass){

		int i = threadIdx.x + blockDim.x * blockIdx.x;
		vel_x[i] = vel_x[i] + timestep_length * 0.5 * (F_x[i] + F_old_x[i])/mass[i];
                vel_y[i] = vel_y[i] + timestep_length * 0.5 * (F_y[i] + F_old_y[i])/mass[i];
                vel_z[i] = vel_z[i] + timestep_length * 0.5 * (F_z[i] + F_old_z[i])/mass[i];

}















