#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<fstream>
#include<math.h>

__global__ void kern_Force(     double *epsilon,
				double *sigma,
				double *d_F_old,
				double *d_F,
				double *d_x){
	////+++++++++++++++++++++++++++++++++++++++++
	int x_index = threadIdx.x + blockIdx.x*blockDim.x;
	int y_index = threadIdx.y + blockIdx.y*blockDim.y;
	int z_index = threadIdx.z + blockIdx.z*blockDim.z;
	
    for(k=0;k<N;++k){
        for(i=0;i<N;++i)
        {
            
            for(j=0;j<N;j++{
                        
                
            }
        
        }
    }

}
int main()
{
	//parameters to be read from the input file
	double timestep_length = 0.01;
	double time_end = 1.0;
	double epsilon = 5.0;
	double sigma = 1.0;
	unsigned int N = 1000;

	//initialization of the variables
	std::vector<double> x;
	x.resize(3*N);
	std::vector<double> v;
	v.resize(3*N);
	std::vector<double> a;
	a.resize(3*N);
	std::vector<double> F;
	F.resize(3*N);
	std::vector<double> F_old;
	
	//reading the data from the input file for test

	//initializing pointers for hipMalloc
	std::vector<double> d_x;
	std::vector<double> d_v;
	std::vector<double> d_a;
	std::vector<double> d_F;
	std::vector<double> d_F_old;
	
	
	
	//kernel for calculation of force
	hipMalloc((void**)&d_x, sizeof(std::vector<double>));
	d_x.resize(N*3);
	hipMalloc((void**)&d_v, sizeof(std::vector<double>));
	d_v.resize(N*3);
	hipMalloc((void**)&d_a, sizeof(std::vector<double>));
	d_a.resize(N*3);
	hipMalloc((void**)&d_F, sizeof(std::vector<double>));
	d_F.resize(3*N);
	hipMalloc((void**)&d_F_old, sizeof(std::vector<double>));
	d_F_old.resize(3*N);

	//memcopy from host to device
	hipMemcpy(d_x,x,3*N*sizeof(std::vector<double>),hipMemcpyHostToDevice);
	hipMemcpy(d_a,a,3*N*sizeof(std::vector<double>),hipMemcpyHostToDevice);
	hipMemcpy(d_v,v,3*N*sizeof(std::vector<double>),hipMemcpyHostToDevice);
	hipMemcpy(d_F,F,3*N*sizeof(std::vector<double>),hipMemcpyHostToDevice);
	hipMemcpy(d_F_old,F_old,3*N*sizeof(std::vector<double>),hipMemcpyHostToDevice);

	//call kern_Force for calculating the force between particles
	

	//call kern_Vel_Verlet for updating the velocities
	
	//memcopy from device back to host
	hipMemcpy(x,d_x,3*N*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(a,d_a,3*N*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(v,d_v,3*N*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(F,d_F,3*N*sizeof(double),hipMemcpyHostToDevice);
        hipMemcpy(F_old,d_F_old,3*N*sizeof(double),hipMemcpyHostToDevice);

	//free device memory after simulation
	hipFree(d_x);
	hipFree(d_v);
	hipFree(d_a);
	hipFree(d_F_old);
	hipFree(d_F);
	

	return 0;
}
